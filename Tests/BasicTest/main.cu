#include "hip/hip_runtime.h"
/* ************************************************************************************************************* */
/*                                                                                                               */
/*     Kartet                                                                                                    */
/*     A Simple C++ Array Library for CUDA                                                                       */
/*                                                                                                               */
/*     LICENSE : The MIT License                                                                                 */
/*     Copyright (c) 2015 Ronan Kerviche                                                                         */
/*                                                                                                               */
/*     Permission is hereby granted, free of charge, to any person obtaining a copy                              */
/*     of this software and associated documentation files (the "Software"), to deal                             */
/*     in the Software without restriction, including without limitation the rights                              */
/*     to use, copy, modify, merge, publish, distribute, sublicense, and/or sell                                 */
/*     copies of the Software, and to permit persons to whom the Software is                                     */
/*     furnished to do so, subject to the following conditions:                                                  */
/*                                                                                                               */
/*     The above copyright notice and this permission notice shall be included in                                */
/*     all copies or substantial portions of the Software.                                                       */
/*                                                                                                               */
/*     THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR                                */
/*     IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,                                  */
/*     FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE                               */
/*     AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER                                    */
/*     LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,                             */
/*     OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN                                 */
/*     THE SOFTWARE.                                                                                             */
/*                                                                                                               */
/* ************************************************************************************************************* */

#include <iostream>
#include "Kartet.hpp"

int main(int argc, char** argv)
{
	UNUSED_PARAMETER(argc)
	UNUSED_PARAMETER(argv)

	int returnCode = 0;
	std::cout << "================================" << std::endl;
	std::cout << "      Kartet Syntax Example     " << std::endl;
	std::cout << "================================" << std::endl;
	std::cout << "Build : " << __DATE__ << ' ' << __TIME__ << std::endl;
	srand(time(NULL));

	try
	{
		// Creating a Layout (size of an array) :
		const Kartet::Layout layout(5, 3);

		// Creating arrays :
		Kartet::Array<double> 	A(layout),
					B(layout),
					C(layout);

		/*Kartet::Array<hipDoubleComplex, Kartet::HostSide> Cx(layout);

		// Initialization, fill with ones :
		A = 1.0;
		std::cout << "A = " << A << std::endl;

		// Initialization with the row indices :
		B = Kartet::IndexI();
		std::cout << "B = " << B << std::endl;

		Cx = Kartet::toComplex(Kartet::IndexI(), Kartet::IndexJ());
		std::cout << "Cx = " << Cx << std::endl;

		// Compute an expression :
		C = 2.0*(B-A);
		std::cout << "C = " << C << std::endl;

		Cx = Cx * Cx - 1.0;
		std::cout << "Cx = " << Cx << std::endl;

		// Computing over parts of the array :
		C.vector(0) = C.vector(2) - C.vector(1);
		std::cout << "C = " << C << std::endl;
		
		Kartet::UniformSource<> uniformSource;
		uniformSource.setSeed();
		uniformSource >> A;
		uniformSource >> B;
		std::cout << "A = " << A << std::endl;
		std::cout << "B = " << B << std::endl;

		// Reduction example :
		Kartet::ReduceContext reduceContext;
		const double count1 = reduceContext.sum(A.getLayout(), Kartet::cast<int>(A>=B));
		std::cout << "Number of elements of A larger than B : " << count1 << std::endl;
		
		const double sum1 = reduceContext.sum(A.getLayout(), abs(A-B));
		std::cout << "Sum over |A-B| : " << sum1 << std::endl;

		// BLAS Example :
		const Kartet::Layout matrixLayout(4,4);
		Kartet::Array<double> M1(matrixLayout), M2(matrixLayout), M3(matrixLayout);
		M1 = Kartet::IndexI();
		M2 = (Kartet::IndexI() + Kartet::IndexJ())/2.0;
		M3 = 0.0;

		Kartet::BLASContext blas;
		blas.gemm(M1,M2,M3);
		std::cout << "M1 = " << M1 << std::endl;
		std::cout << "M2 = " << M2 << std::endl;
		std::cout << "M3 = " << M3 << std::endl;*/
	}
	catch(Kartet::Exception& e)
	{
		std::cout << "Exception : " << e << std::endl;
		returnCode = -1;
	}

	return returnCode;
}

