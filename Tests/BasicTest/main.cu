#include "hip/hip_runtime.h"
/* ************************************************************************************************************* */
/*                                                                                                               */
/*     Kartet                                                                                                    */
/*     A Simple C++ Array Library for CUDA                                                                       */
/*                                                                                                               */
/*     LICENSE : The MIT License                                                                                 */
/*     Copyright (c) 2015 Ronan Kerviche                                                                         */
/*                                                                                                               */
/*     Permission is hereby granted, free of charge, to any person obtaining a copy                              */
/*     of this software and associated documentation files (the "Software"), to deal                             */
/*     in the Software without restriction, including without limitation the rights                              */
/*     to use, copy, modify, merge, publish, distribute, sublicense, and/or sell                                 */
/*     copies of the Software, and to permit persons to whom the Software is                                     */
/*     furnished to do so, subject to the following conditions:                                                  */
/*                                                                                                               */
/*     The above copyright notice and this permission notice shall be included in                                */
/*     all copies or substantial portions of the Software.                                                       */
/*                                                                                                               */
/*     THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR                                */
/*     IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,                                  */
/*     FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE                               */
/*     AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER                                    */
/*     LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,                             */
/*     OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN                                 */
/*     THE SOFTWARE.                                                                                             */
/*                                                                                                               */
/* ************************************************************************************************************* */

#include <iostream>
#include "Kartet.hpp"

	__global__ void testPrint(const Kartet::Layout layout)
	{
		const Kartet::index_t	i = layout.getI(), 
					j = layout.getJ(), 
					k = layout.getK(),
					p = layout.getIndex();
		printf("  Hi, from block : (%d; %d; %d) Thread : (%d; %d; %d) => (i=%d; j=%d; k=%d):p=%d\n", blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z, (int)i, (int)j, (int)k, (int)p);
	}

int main(int argc, char** argv)
{
	int returnCode = 0;
	std::cout << "================================" << std::endl;
	std::cout << "       Kartet Syntax Tests      " << std::endl;
	std::cout << "================================" << std::endl;
	std::cout << "Build : " << __DATE__ << ' ' << __TIME__ << std::endl;
	srand(time(NULL));

	try
	{
		// Testing expressions :
		Kartet::Array<double> A(5, 3), B(5, 3);

		A = Kartet::IndexI();
		B = Kartet::IndexJ();
		std::cout << "A = " << A << std::endl;
		std::cout << "B = " << B << std::endl;
		A = A+B;
		std::cout << "S = " << A << std::endl;
		
		std::cout << "Layout : " << A.getLayout() << std::endl;
		A = 1.23;
		B = 2.56 * Kartet::IndexJ();
		A = cos(A + exp(2.0 * B));
		std::cout << "A = " << A << std::endl;

		hipDeviceSynchronize();
		std::cout << "Test kernel : " << std::endl;
		testPrint COMPUTE_LAYOUT(A) (A.getLayout());
		hipDeviceSynchronize();
		std::cout << "Done." << std::endl;

		A = min(A, B);
		std::cout << "A = " << A << std::endl;

		Kartet::Array<double> C(16);
		C = Kartet::IndexI();
		std::cout << "C = " << C << std::endl;

		C = Kartet::cast<unsigned char>(Kartet::IndexI()*64);
		std::cout << "C = " << C << std::endl;

		// BLAS :
		Kartet::BLASContext blas;
		C = 4 - absSq(Kartet::IndexI()-13) / 100.0;
		std::cout << "C = " << C << std::endl;
		int idx = blas.amax(C);
		std::cout << "Index of the absolute maximum : " << idx << std::endl;
		std::cout << std::endl;

		const int w1 = 256, w2 = 512, w3 = 348;
		Kartet::Array<float> X(w1, w3), Y(w3, w2), Z(w1, w2);
		X = Kartet::IndexI();
		Y = Kartet::IndexJ();
		Z = 0.0;
		//gemm(1.0, X, HIPBLAS_OP_N, Y, HIPBLAS_OP_N, 0.0, Z);
		blas.gemm(X, Y, Z);

		// Generate random numbers :
		Kartet::RandomSourceContext randomSourceContext;
		randomSourceContext.setSeed();
		Kartet::UniformSource uniformSource;
		uniformSource >> A;
		std::cout << "A = " << A << std::endl;

		// Accessing data :
		A = Kartet::IndexI() + Kartet::IndexJ();
		// Select 2 vectors starting at 0 with a step of 2 (1st and 3rd vectors).
		Kartet::Accessor<double> S = A.vectors(0, 2, 2);
		uniformSource >> S;
		std::cout << "S = " << S << std::endl;
		std::cout << "A = " << A << std::endl;
		A.subArray(1,0,3,3) = -23.0;
		std::cout << A << std::endl;
	
		// More manipulations with accessors :
		Kartet::Array<float> D(4, 4, 3);
		Kartet::NormalSource normalSource(2.0, 10.0);
		normalSource >> D;
		std::cout << "D = " << D << std::endl;
		D.vectors(0, 2, 2) = 0;
		std::cout << "D = " << D << std::endl;
		D.vectors(0, 2, 2).slice(1) = 1.0;
		std::cout << "D = " << D << std::endl;

		// Computing on complex numbers without storing :
		D = real(piAngleToComplex(Kartet::IndexI() + Kartet::IndexJ()));
		std::cout << "D = " << D << std::endl;
		std::cout << "Layout of D.slices(0, 2, 2) : " << D.slices(0, 2, 2).getLayout() << std::endl;
		D.slice(0) = D.slice(0)*D.slice(1);
		std::cout << "D.slice(0) = " << D.slice(0) << std::endl;
		D.slice(1) = (D.slice(1) + D.slice(0))/2.0;
		std::cout << "D = " << D << std::endl;

		// Complex :
		Kartet::Array<hipDoubleComplex> CxA(4, 4);
		CxA = 1.0 + Kartet::IndexJ();
		CxA = angleToComplex(real(CxA));
		std::cout << "CxA = " << CxA << std::endl;
		Kartet::Array<double> CxAbs(CxA.getLayout());
		CxAbs = abs(CxA) - real(CxA);
		std::cout << "CxAbs = " << CxAbs << std::endl;
		CxAbs = angle(CxA);
		std::cout << "CxAbs = " << CxAbs << std::endl;

		// File I/O :
		Kartet::Array<int> U(8, 8);
		U = Kartet::IndexI() + Kartet::IndexJ();
		U.writeToFile("tmp.dat");
		std::cout << "U = " << U << std::endl;
		Kartet::Array<float> V(8, 8);
		V.readFromFile("tmp.dat");
		std::cout << "V = " << V << std::endl;
		Kartet::Array<hipDoubleComplex> W("tmp.dat");
		std::cout << "W = " << W << std::endl;

		// Reduction :
		Kartet::ReduceContext reduceContext;
		const Kartet::Layout l(4661,7965);
		std::cout << "Layout : " << l << std::endl;
		int sum1 = reduceContext.sum(l, 1);
		std::cout << "Sum(1) : " << sum1 << " == " << l.getNumElements() << ", test : " << (sum1==l.getNumElements()) << std::endl;

		double  sum2 = reduceContext.sum(l, Kartet::cast<double>(Kartet::IndexI()+Kartet::IndexJ())),
			res2 = static_cast<double>(l.getNumRows()+l.getNumColumns()-2)*static_cast<double>(l.getNumRows()*l.getNumColumns())/2.0;
		std::cout << "Sum(I()+J()) : " << sum2 << " == " << res2 << ", diff : " << std::abs(sum2-res2)/res2 << std::endl;
		
		int testReduction = reduceContext.sum(U);
		std::cout << "Sum(U) : " << testReduction << std::endl;
		testReduction = reduceContext.sum(U.getLayout(), Kartet::IndexI()*Kartet::IndexJ());
		std::cout << "Sum(I()*J()) = " << testReduction << " (==784 on 8x8)" << std::endl;
		std::cout << "Sum(sqrt(I()*J())) = " << reduceContext.sum(Kartet::Layout(16, 16), sqrt(Kartet::cast<double>(Kartet::IndexI()*Kartet::IndexJ()))) << " (\\approx 1637.755873, for 16x16)" << std::endl;
	}
	catch(Kartet::Exception& e)
	{
		std::cout << "Exception : " << e << std::endl;
		returnCode = -1;
	}

	return returnCode;
}

